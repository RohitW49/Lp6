#include <iostream>
#include <hip/hip_runtime.h>

__global__ void addVectors(int *A, int *B, int *C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int n = 1000000;
    const int blockSize = 256;
    const int numBlocks = (n + blockSize - 1) / blockSize;

    // Host arrays
    int *A, *B, *C;
    hipHostMalloc(&A, n * sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&B, n * sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&C, n * sizeof(int), hipHostMallocDefault);

    // Initialize vectors A and B
    for (int i = 0; i < n; ++i) {
        A[i] = i;
        B[i] = i * 2;
    }

    // Device arrays
    int *dev_A, *dev_B, *dev_C;
    hipMalloc(&dev_A, n * sizeof(int));
    hipMalloc(&dev_B, n * sizeof(int));
    hipMalloc(&dev_C, n * sizeof(int));

    // Copy data from host to device
    hipMemcpy(dev_A, A, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    addVectors<<<numBlocks, blockSize>>>(dev_A, dev_B, dev_C, n);
    hipDeviceSynchronize();

    // Copy result from device to host
    hipMemcpy(C, dev_C, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    for (int i = 0; i < 10; ++i) {
        std::cout << C[i] << " ";
    }
    std::cout << std::endl;

    // Free memory
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);

    return 0;
}
