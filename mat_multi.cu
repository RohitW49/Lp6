#include <iostream>
#include <hip/hip_runtime.h>

const int N = 1024; // Matrix size (N x N)

// Kernel function to perform matrix multiplication
__global__ void matrixMul(float *A, float *B, float *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < n; ++i) {
            sum += A[row * n + i] * B[i * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    // Allocate host memory and initialize matrices A and B with random values
    float *h_A = new float[N * N];
    float *h_B = new float[N * N];
    float *h_C = new float[N * N];
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = rand() / RAND_MAX;
        h_B[i] = rand() / RAND_MAX;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * N * sizeof(float));

    // Copy input matrices from host to device
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 blockSize(32, 32);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);
    matrixMul<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

    // Copy result matrix from device to host
    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        for (int j = 0; j < 10; j++) {
            std::cout << h_C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }
    // Free memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
